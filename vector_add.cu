
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

// Kernel function to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000;
    int size = n * sizeof(int);
    
    // Allocate memory on the host
    int *h_a = (int *)malloc(size);
    int *h_b = (int *)malloc(size);
    int *h_c = (int *)malloc(size);

    // Initialize vectors
    for(int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Allocate memory on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    for(int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

